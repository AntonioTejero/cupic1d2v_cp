#include "hip/hip_runtime.h"
/****************************************************************************
 *                                                                          *
 *    CUPIC1D is a code that simulates the interaction between plasma and   *
 *    a langmuir probe in 1D using PIC techniques accelerated with the use  * 
 *    of GPU hardware (CUDA, extension of C/C++)                            *
 *                                                                          *
 ****************************************************************************/


/****************************** HEADERS ******************************/

#include "particles.h"

/********************* HOST FUNCTION DEFINITIONS *********************/

void particle_mover(particle *d_i, int num_i, double *d_E) 
{
  /*--------------------------- function variables -----------------------*/
  
  // host memory
  static const double me = init_me();     // electron's mass
  static const double mi = init_mi();     // ion's mass
  static const double qe = init_qe();     // electron's charge
  static const double qi = init_qi();     // ions's charge
  static const double ds = init_ds();     // spatial step
  static const double r_p = init_r_p();   // spatial step
  static const double dt = init_dt();     // time step
  static const int nn = init_nn();        // number of nodes  
  
  dim3 griddim, blockdim;
  size_t sh_mem_size;
  
  // device memory
  
  /*----------------------------- function body -------------------------*/
  
  // set size of __shared__ memory for leap_frog kernel
  sh_mem_size = nn*sizeof(double);

  //---- move ions  
 
  // set dimensions of grid of blocks and blocks of threads for leap_frog kernel
  blockdim = PAR_MOV_BLOCK_DIM;
  griddim = int(num_i/PAR_MOV_BLOCK_DIM)+1;
 
  // call to leap_frog_step kernel (ions)
  hipGetLastError();
  leap_frog_step<<<griddim, blockdim, sh_mem_size>>>(qi, mi, num_i, d_i, dt, ds, r_p, nn, d_E);
  cu_sync_check(__FILE__, __LINE__);
  
  return;
}

/**********************************************************/



/******************** DEVICE KERNELS DEFINITIONS *********************/

__global__ void leap_frog_step(double q, double m, int num_p, particle *g_p, double dt, double ds, 
                               double r_p, int nn, double *g_E)
{
  /*--------------------------- kernel variables -----------------------*/
  
  // kernel shared memory
  double *sh_E = (double *) sh_mem;   // manually set up shared memory variables

  // kernel registers
  int tidx = (int) threadIdx.x;
  int tid = (int) threadIdx.x + (int) blockDim.x * (int) blockIdx.x;  // thread Id
  int bdim = (int) blockDim.x;  // block dimension
  particle reg_p;               // register particles
  int ic;                       // cell index
  double dist;                  // distance from particle to nearest down vertex (normalized to ds)
  double F;                     // force suffered for each register particle
  double dummy_r;               // intermediate new position

  /*--------------------------- kernel body ----------------------------*/
  
  //---- initialize shared memory variables
  
  // load fields from global memory
  for (int i = tidx; i<nn; i += bdim) {
    sh_E[i] = g_E[i];
  }
  __syncthreads();
  
  //---- Process batches of particles
  
  if (tid < num_p) {
    // load particle data in registers
    reg_p = g_p[tid];
    
    // find cell index
    ic = __double2int_rd(reg_p.r/ds);

    // evaluate distance to nearest down vertex (normalized to ds)
    dist = fabs(reg_p.r-ic*ds)/ds;

    // calculate particle's forces
    F = q*(sh_E[ic]*(1.0-dist) + sh_E[ic+1]*dist)+m*reg_p.vt*reg_p.vt/(reg_p.r+r_p);

    // move particles
    reg_p.vr += dt*F/m;
    dummy_r = reg_p.r + dt*reg_p.vr;
    reg_p.vt *= (reg_p.r+r_p)/(dummy_r+r_p);
    reg_p.r = dummy_r;
    
    // store particle data in global memory
    g_p[tid] = reg_p;
  }
  
  return;
}

/**********************************************************/


/******************** DEVICE FUNCTION DEFINITIONS ********************/

